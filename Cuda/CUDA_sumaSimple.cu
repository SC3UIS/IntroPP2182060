
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel que realiza la suma en paralelo
__global__ void sum_kernel(int *sum, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index <= num && index > 0) {
        atomicAdd(sum, index);
    }
}

int main() {
    int num = 10;
    int sum = 0;

    // Alocamos memoria en la GPU para la suma
    int *dev_sum;
    hipMalloc((void **)&dev_sum, sizeof(int));

    // Copiamos la suma desde la memoria de la CPU a la de la GPU
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

    // Creamos los eventos para medir el tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Empezamos a medir el tiempo
    hipEventRecord(start);

    // Lanzamos el kernel en la GPU
    sum_kernel<<<num + 1, 1>>>(dev_sum, num);

    // Detenemos el tiempo despu�s de que el kernel termine
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time: %f ms\n", milliseconds);
    printf("Throughput: %f operations/ms\n", num / milliseconds);

    // Copiamos la suma desde la memoria de la GPU a la de la CPU
    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSum = %d\n", sum);

    // Liberamos la memoria de la GPU
    hipFree(dev_sum);

    return 0;
}
